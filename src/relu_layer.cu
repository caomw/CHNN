#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "../include/relu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : 0;
  }
}

template <typename Dtype>
void ReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

INSTANTIATE_LAYER_GPU_FUNCS(ReLULayer);
}  // namespace caffe
